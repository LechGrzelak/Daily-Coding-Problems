

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c){
  *c = *a + *b  ;
}

int main(){
  int a, b, c;
  int *d_a, *d_b, *d_c;
  int size = sizeof(int);

  // int **p1 = &d_a;
  // int **p2 = &d_b;
  // int **p3 = &d_c;
  // cudaMalloc(p1, size);
  // cudaMalloc(p2, size);
  // cudaMalloc(p3, size);
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  a = 10; b = 20;

  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
  // cudaMemcpy(d_c, &c, size, cudaMemcpyHostToDevice);
  add<<<1,1>>>(d_a, d_b, d_c);
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  printf("SUM=%d\n", c);

  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
