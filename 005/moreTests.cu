#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a, int *b, int *c, char *words_d){
  *c = 100;
  *words_d = "i am a good string";
  printf("how to print this\n");
}

int main(){
  int a, b, c;
  int *d_a, *d_b, *d_c;
  int size = sizeof(int);
  char w;
  char *w_d;

  // int **p1 = &d_a;
  // int **p2 = &d_b;
  // int **p3 = &d_c;
  // hipMalloc(p1, size);
  // hipMalloc(p2, size);
  // hipMalloc(p3, size);
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  hipMalloc((void **)&d_c, sizeof(char));
  a = 10; b = 20;

  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
  // hipMemcpy(d_c, &c, size, hipMemcpyHostToDevice);
  add<<<1,1>>>(d_a, d_b, d_c);
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  printf("SUM=%d\n", c);

  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
