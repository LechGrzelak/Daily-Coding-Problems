
// program to return a constant value from device

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void test(int *n){
  *n = 0;
}

int main(){
  int x, *d_x;
  int size = sizeof(int);
  hipMalloc((void **)&d_x, size);
  test<<<1,1>>>(d_x);
  hipMemcpy(&x, d_x, size, hipMemcpyDeviceToHost);
  printf("returned from device = %d\n", x);
  return 0;
}
